//
// NOTE: compile this .cu module for compute_35,sm_35 with --maxrregcount=64
//

#include "hip/hip_runtime.h"



typedef unsigned int uint32_t; // define this as 32 bit type derived from int

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <map>
#include <algorithm>

#include <stdbool.h>

// Define work unit size
#define WU_PER_WARP 32
#define WU_PER_BLOCK (WU_PER_WARP*WARPS_PER_BLOCK)
#define WU_PER_LAUNCH (GRID_BLOCKS*WU_PER_BLOCK)
#define SCRATCH (32768+64)

// Not performing error checking is actually bad, but...
#define checkCudaErrors(x) x
#define getLastCudaError(x)

// from salsa_kernel.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

__constant__ uint32_t* c_V[1024];

#define ROTL(a, b) __funnelshift_l( a, a, b );

static __device__ __forceinline__ void lock(int *mutex, int i)
{
    while( atomicCAS( &mutex[i], 0, 1 ) != 0 );
}

static __device__ __forceinline__ void unlock(int *mutex, int i)
{
    atomicExch( &mutex[i], 0 );
}

static __device__ __forceinline__ void xor_salsa8(uint32_t *B, const uint32_t *C)
{
    uint32_t x0 = (B[ 0] ^= C[ 0]), x1 = (B[ 1] ^= C[ 1]), x2 = (B[ 2] ^= C[ 2]), x3 = (B[ 3] ^= C[ 3]);
    uint32_t x4 = (B[ 4] ^= C[ 4]), x5 = (B[ 5] ^= C[ 5]), x6 = (B[ 6] ^= C[ 6]), x7 = (B[ 7] ^= C[ 7]);
    uint32_t x8 = (B[ 8] ^= C[ 8]), x9 = (B[ 9] ^= C[ 9]), xa = (B[10] ^= C[10]), xb = (B[11] ^= C[11]);
    uint32_t xc = (B[12] ^= C[12]), xd = (B[13] ^= C[13]), xe = (B[14] ^= C[14]), xf = (B[15] ^= C[15]);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    B[ 0] += x0; B[ 1] += x1; B[ 2] += x2; B[ 3] += x3; B[ 4] += x4; B[ 5] += x5; B[ 6] += x6; B[ 7] += x7;
    B[ 8] += x8; B[ 9] += x9; B[10] += xa; B[11] += xb; B[12] += xc; B[13] += xd; B[14] += xe; B[15] += xf;
}

static __device__ __forceinline__ uint2& operator^=(uint2& left, const uint2& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    return left;
}

////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel with spinlock guards around a smaller shared memory
//! Version for Geforce Titan, low register count (<=64), low shared mem use.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
scrypt_core_kernel_spinlock_titanA(uint32_t *g_idata, int *mutex)
{
    volatile __shared__ uint32_t X[WU_PER_WARP][16+2]; // +2 to reduce bank conflicts
                                                       // while maintaining alignment
    int warpIdx         = threadIdx.x / warpSize;
    int warpThread      = threadIdx.x % warpSize;
    int WARPS_PER_BLOCK = blockDim.x / warpSize;

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_idata += 32 * offset;
    uint32_t* V = (uint32_t*)c_V[offset/WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/8;
    volatile unsigned int Z = 2*(warpThread%8);

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    if (warpThread == 0) lock(mutex, blockIdx.x);
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&V[SCRATCH*(wu+Y)+Z])) = *((uint2*)(&X[wu+Y][Z])) = *((uint2*)(&g_idata[32*(wu+Y)+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpThread][idx];

#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&V[SCRATCH*(wu+Y)+16+Z])) = *((uint2*)(&X[wu+Y][Z])) = *((uint2*)(&g_idata[32*(wu+Y)+16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpThread][idx];

    for (int i = 1; i < 1024; i++) {

        if (warpThread == 0) unlock(mutex, blockIdx.x);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x);

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = B[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&V[SCRATCH*(wu+Y) + i*32 + Z])) = *((uint2*)(&X[wu+Y][Z]));

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = C[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&V[SCRATCH*(wu+Y) + i*32 + 16 + Z])) = *((uint2*)(&X[wu+Y][Z]));
    }
    if (warpThread == 0) unlock(mutex, blockIdx.x);
}

__global__ void
scrypt_core_kernel_spinlock_titanB(uint32_t *g_odata, int *mutex)
{
    volatile __shared__ uint32_t X[WU_PER_WARP][16+2]; // +2 to reduce bank conflicts
                                                       // while maintaining alignment
    int warpIdx         = threadIdx.x / warpSize;
    int warpThread      = threadIdx.x % warpSize;
    int WARPS_PER_BLOCK = blockDim.x / warpSize;

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32 * offset;
    const uint32_t* __restrict__ V = (const uint32_t*)c_V[offset/WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/8;
    volatile unsigned int Z = 2*(warpThread%8);

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    if (warpThread == 0) lock(mutex, blockIdx.x);
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&X[wu+Y][Z])) = *((uint2*)(&V[SCRATCH*(wu+Y) + 1023*32 + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpThread][idx];

#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&X[wu+Y][Z])) = *((uint2*)(&V[SCRATCH*(wu+Y) + 1023*32 + 16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpThread][idx];

    if (warpThread == 0) unlock(mutex, blockIdx.x);
    xor_salsa8(B, C); xor_salsa8(C, B);
    if (warpThread == 0) lock(mutex, blockIdx.x);

    for (int i = 0; i < 1024; i++) {

        X[warpThread][16] = C[0];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = B[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&X[wu+Y][Z])) ^= *((uint2*)(&V[SCRATCH*(wu+Y) + 32*(X[wu+Y][16] & 1023) + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpThread][idx];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = C[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&X[wu+Y][Z])) ^= *((uint2*)(&V[SCRATCH*(wu+Y) + 32*(X[wu+Y][16] & 1023) + 16 + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpThread][idx];

        if (warpThread == 0) unlock(mutex, blockIdx.x);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x);
    }

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = B[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&g_odata[32*(wu+Y)+Z])) = *((uint2*)(&X[wu+Y][Z]));

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpThread][idx] = C[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&g_odata[32*(wu+Y)+16+Z])) = *((uint2*)(&X[wu+Y][Z]));

    if (warpThread == 0) unlock(mutex, blockIdx.x);
}

////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel
//! Version for Geforce Titan, low register count (<=64), low shared mem use.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int WARPS_PER_BLOCK> __global__ void
scrypt_core_kernel_titanA(uint32_t *g_idata)
{
    volatile __shared__ uint32_t X[WARPS_PER_BLOCK][WU_PER_WARP][16+2]; // +2 to reduce bank conflicts
                                                                        // while maintaining alignment
    int warpIdx        = threadIdx.x / warpSize;
    int warpThread     = threadIdx.x % warpSize;

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_idata += 32 * offset;
    uint32_t* V = (uint32_t*)c_V[offset/WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/8;
    volatile unsigned int Z = 2*(warpThread%8);

    // registers to store an entire work unit
    uint32_t B[16], C[16];

#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&V[SCRATCH*(wu+Y)+Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z])) = *((uint2*)(&g_idata[32*(wu+Y)+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx][warpThread][idx];

#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&V[SCRATCH*(wu+Y)+16+Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z])) = *((uint2*)(&g_idata[32*(wu+Y)+16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx][warpThread][idx];

    for (int i = 1; i < 1024; i++) {

        xor_salsa8(B, C); xor_salsa8(C, B);

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx][warpThread][idx] = B[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&V[SCRATCH*(wu+Y) + i*32 + Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z]));

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx][warpThread][idx] = C[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&V[SCRATCH*(wu+Y) + i*32 + 16 + Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z]));
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel
//! Version for Geforce Titan, low register count (<=64), low shared mem use.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int WARPS_PER_BLOCK> __global__ void
scrypt_core_kernel_titanB(uint32_t *g_odata)
{
    volatile __shared__ uint32_t X[WARPS_PER_BLOCK][WU_PER_WARP][16+2]; // +2 to reduce bank conflicts
                                                                        // while maintaining alignment
    int warpIdx        = threadIdx.x / warpSize;
    int warpThread     = threadIdx.x % warpSize;

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32 * offset;
    const uint32_t* __restrict__ V = (const uint32_t*)c_V[offset/WU_PER_WARP];

    // variables supporting the large memory transaction magic
    volatile unsigned int Y = warpThread/8;
    volatile unsigned int Z = 2*(warpThread%8);

    // registers to store an entire work unit
    uint32_t B[16], C[16];

#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&X[warpIdx][wu+Y][Z])) = *((uint2*)(&V[SCRATCH*(wu+Y) + 1023*32 + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx][warpThread][idx];

#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&X[warpIdx][wu+Y][Z])) = *((uint2*)(&V[SCRATCH*(wu+Y) + 1023*32 + 16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx][warpThread][idx];

    xor_salsa8(B, C); xor_salsa8(C, B);

    for (int i = 0; i < 1024; i++) {

        X[warpIdx][warpThread][16] = C[0];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx][warpThread][idx] = B[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&X[warpIdx][wu+Y][Z])) ^= *((uint2*)(&V[SCRATCH*(wu+Y) + 32*(X[warpIdx][wu+Y][16] & 1023) + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx][warpThread][idx];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx][warpThread][idx] = C[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
            *((uint2*)(&X[warpIdx][wu+Y][Z])) ^= *((uint2*)(&V[SCRATCH*(wu+Y) + 32*(X[warpIdx][wu+Y][16] & 1023) + 16 + Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx][warpThread][idx];

        xor_salsa8(B, C); xor_salsa8(C, B);
    }

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpIdx][warpThread][idx] = B[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&g_odata[32*(wu+Y)+Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z]));

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpIdx][warpThread][idx] = C[idx];
#pragma unroll 8
    for (int wu=0; wu < 32; wu+=4)
        *((uint2*)(&g_odata[32*(wu+Y)+16+Z])) = *((uint2*)(&X[warpIdx][wu+Y][Z]));
}


void set_titan_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool run_titan_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, int *mutex, bool special, bool interactive, bool benchmark)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // First phase: Sequential writes to scratchpad.

    if (special)
        scrypt_core_kernel_spinlock_titanA<<< grid, threads, 0, stream >>>(d_idata, mutex);
    else
        switch (WARPS_PER_BLOCK) {
            case 1: scrypt_core_kernel_titanA<1><<< grid, threads, 0, stream >>>(d_idata); break;
            case 2: scrypt_core_kernel_titanA<2><<< grid, threads, 0, stream >>>(d_idata); break;
            case 3: scrypt_core_kernel_titanA<3><<< grid, threads, 0, stream >>>(d_idata); break;
            case 4: scrypt_core_kernel_titanA<4><<< grid, threads, 0, stream >>>(d_idata); break;
            case 5: scrypt_core_kernel_titanA<5><<< grid, threads, 0, stream >>>(d_idata); break;
            case 6: scrypt_core_kernel_titanA<6><<< grid, threads, 0, stream >>>(d_idata); break;
            case 7: scrypt_core_kernel_titanA<7><<< grid, threads, 0, stream >>>(d_idata); break;
            case 8: scrypt_core_kernel_titanA<8><<< grid, threads, 0, stream >>>(d_idata); break;
            default: success = false; break;
        }

    // Optional millisecond sleep in between kernels

    if (!benchmark && interactive) {
        checkCudaErrors(MyStreamSynchronize(stream, 1, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    // Second phase: Random read access from scratchpad.

    if (special)
        scrypt_core_kernel_spinlock_titanB<<< grid, threads, 0, stream >>>(d_odata, mutex);
    else
        switch (WARPS_PER_BLOCK) {
            case 1: scrypt_core_kernel_titanB<1><<< grid, threads, 0, stream >>>(d_odata); break;
            case 2: scrypt_core_kernel_titanB<2><<< grid, threads, 0, stream >>>(d_odata); break;
            case 3: scrypt_core_kernel_titanB<3><<< grid, threads, 0, stream >>>(d_odata); break;
            case 4: scrypt_core_kernel_titanB<4><<< grid, threads, 0, stream >>>(d_odata); break;
            case 5: scrypt_core_kernel_titanB<5><<< grid, threads, 0, stream >>>(d_odata); break;
            case 6: scrypt_core_kernel_titanB<6><<< grid, threads, 0, stream >>>(d_odata); break;
            case 7: scrypt_core_kernel_titanB<7><<< grid, threads, 0, stream >>>(d_odata); break;
            case 8: scrypt_core_kernel_titanB<8><<< grid, threads, 0, stream >>>(d_odata); break;
            default: success = false; break;
        }

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}
